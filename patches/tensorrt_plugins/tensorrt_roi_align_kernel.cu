#include "common_cuda_helper.h"
#include "roi_align_cuda_kernel.cuh"

template <typename scalar_t>
void RoIAlignForwardCUDAKernelLauncher( const scalar_t* bottom_data
                                      , const scalar_t spatial_scale
                                      , const int output_size
                                      , const int channels
                                      , const int height
                                      , const int width
                                      , const int pooled_height
                                      , const int pooled_width
                                      , const int sampling_ratio
                                      , const scalar_t* bottom_rois
                                      , const int roi_cols
                                      , scalar_t* top_data
                                      , const int is_mode_avg
                                      , const int* batch_indices_ptr
                                      , hipStream_t stream) {
  RoIAlignForward<scalar_t>
      <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>( output_size // nthreads
                                                                 , bottom_data // bottom_data
                                                                 , spatial_scale // spatial_scale
                                                                 , channels // channels
                                                                 , height // height
                                                                 , width // width
                                                                 , pooled_height // pooled_height
                                                                 , pooled_width // pooled_width
                                                                 , sampling_ratio // sampling_ratio
                                                                 , bottom_rois // bottom_rois
                                                                 , roi_cols // roi_cols
                                                                 , top_data // top_data
                                                                 , is_mode_avg // is_mode_avg
                                                                 , batch_indices_ptr); // batch_indices_ptr
}

void RoIAlignForwardCUDAKernelLauncher_float( const float* bottom_data
                                            , const float spatial_scale
                                            , const int output_size
                                            , const int channels
                                            , const int height
                                            , const int width
                                            , const int pooled_height
                                            , const int pooled_width
                                            , const int sampling_ratio
                                            , const float* bottom_rois
                                            , const int roi_cols
                                            , float* top_data
                                            , const int is_mode_avg
                                            , const int* batch_indices_ptr
                                            , hipStream_t stream) {
  RoIAlignForwardCUDAKernelLauncher<float>( bottom_data // bottom_data
                                          , spatial_scale // spatial_scale
                                          , output_size // output_size
                                          , channels // channels
                                          , height // height
                                          , width // width
                                          , pooled_height // pooled_height
                                          , pooled_width // pooled_width
                                          , sampling_ratio // sampling_ratio
                                          , bottom_rois // bottom_rois
                                          , roi_cols // roi_cols
                                          , top_data // top_data
                                          , is_mode_avg // is_mode_avg
                                          , batch_indices_ptr // batch_indices_ptr
                                          , stream); // streaam
}
